#include <stdio.h>
#include <hip/hip_runtime.h>


int main (int agrc, char **argv) {

  // define the number of data
  int nElem = 1024;

  // define grid and block structure
  dim3 block(1024);
  dim3 grid ((nElem+block.x-1)/block.x);
  printf("grid.x %d block.x %d \n", grid.x, block.x);

  // reset block
  block.x = 512;
  grid.x = (nElem+block.x-1)/block.x;
  printf("grid.x %d block.x %d \n", grid.x, block.x);

  // reset block
  block.x = 256;
  grid.x = (nElem+block.x-1)/block.x;
  printf("grid.x %d block.x %d \n", grid.x, block.x);

  // reset block
  block.x = 128;
  grid.x  = (nElem+block.x)/block.x;
  printf("grid.x %d block.x %d \n", grid.x, block.x);

  // then I reset device before leaving
  hipDeviceReset();

  return(0);
  
}
